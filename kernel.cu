﻿/*

Building process:
CUDA arch and gencode values: https://arnon.dk/matching-sm-architectures-arch-and-gencode-for-various-nvidia-cards/


Perfomance:

THREADS_COUNT = 1024  // 1024 is limit on my GPU (can be higher?)
BLOCK_COUNT = 220  // amount of memory is the only limit here...

Each block has THREADS_COUNT threads.
One kernel (GPU device) invocation has BLOCK_COUNT * THREADS_COUNT parallel workers

NOTE: BLOCK_COUNT can be higher because right now my garbage code isn't super efficient, 
so I'm running out of memory. 

CONTINUE:


Useful links for later:
https://stackoverflow.com/questions/33245737/cuda-estimating-threads-per-blocks-and-block-numbers-for-2d-grid-data/33247118#33247118
*/

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

// FUTURE TODO: Do all sorts of crazy optimazations (__restrict__, branchless and other crazy stuff)
// TODO: explain

typedef unsigned char U8;
typedef unsigned short U16;
typedef unsigned int U32;
typedef unsigned long long int U64;

const int THREADS_COUNT = 1024;  // 1024 <-- limit (can be changed, but that's probably a bad idea)
const int BLOCK_COUNT = 1;  // 4096 * 256 <-- stable. AMOUNT OF MEMORY IS THE ONLY LIMIT
constexpr auto OUTPUT_BUFFER_LEN = THREADS_COUNT * BLOCK_COUNT;

// Bruteforcer params
constexpr auto MAX_SKIP = 26;
constexpr auto L2A_SAFE_CHARS_COUNT = 14;  // 14 for DATA5 using key of length 27 with maximum skips
constexpr auto ITERATIONS_PER_INVOCATION = 32;
constexpr auto KEY_LEN = 7;
constexpr auto PASSING_SCORE = 1;  // used with >
// DATA params
constexpr auto ACTUAL_DATA_SIZE = 511;
constexpr auto ALIGNED_DATA_LEN = 512;

const __constant__ U8 DATA4[ALIGNED_DATA_LEN] = { 73, 118, 84, 117, 105, 116, 110, 32, 116, 97, 116, 32, 32, 101, 72, 101, 110, 111, 69, 105, 32, 112, 103, 118, 105, 32, 73, 65, 84, 102, 32, 110, 65, 79, 105, 77, 32, 112, 76, 108, 32, 115, 115, 121, 71, 110, 101, 108, 110, 117, 32, 108, 40, 69, 101, 111, 32, 115, 101, 73, 116, 104, 72, 78, 99, 59, 89, 65, 82, 115, 76, 32, 111, 101, 101, 111, 69, 105, 108, 110, 66, 115, 116, 40, 32, 101, 108, 116, 104, 32, 100, 121, 108, 97, 114, 120, 116, 108, 83, 65, 80, 97, 32, 89, 98, 83, 112, 78, 82, 39, 85, 114, 115, 108, 67, 80, 116, 104, 59, 66, 32, 76, 105, 121, 82, 79, 115, 111, 77, 112, 115, 70, 32, 83, 114, 97, 115, 67, 73, 115, 115, 110, 32, 75, 78, 69, 65, 76, 104, 65, 84, 105, 104, 32, 111, 32, 32, 97, 116, 111, 119, 114, 115, 116, 122, 67, 110, 105, 115, 117, 104, 99, 65, 72, 75, 65, 109, 32, 32, 101, 99, 73, 82, 69, 102, 116, 78, 115, 85, 77, 105, 78, 110, 97, 110, 32, 72, 101, 111, 115, 105, 114, 104, 72, 83, 115, 59, 65, 100, 116, 117, 117, 78, 112, 111, 100, 112, 69, 111, 59, 112, 101, 121, 32, 114, 54, 84, 32, 41, 114, 115, 40, 105, 115, 103, 32, 116, 121, 65, 105, 59, 115, 115, 101, 59, 116, 70, 101, 67, 116, 117, 116, 69, 106, 32, 83, 111, 101, 119, 117, 111, 101, 32, 116, 77, 78, 82, 32, 66, 32, 83, 101, 100, 99, 108, 32, 67, 32, 75, 120, 101, 101, 67, 79, 120, 97, 105, 59, 32, 116, 110, 105, 116, 76, 108, 76, 121, 32, 116, 101, 116, 110, 99, 85, 32, 79, 79, 32, 105, 106, 119, 69, 105, 67, 58, 98, 55, 79, 59, 101, 85, 105, 116, 104, 120, 32, 86, 79, 111, 100, 65, 114, 32, 41, 84, 117, 32, 120, 98, 115, 99, 58, 111, 110, 68, 32, 78, 110, 32, 116, 112, 115, 111, 116, 32, 75, 83, 101, 111, 59, 97, 101, 41, 68, 77, 69, 119, 50, 122, 32, 116, 102, 32, 69, 121, 102, 111, 116, 100, 97, 110, 114, 65, 101, 101, 116, 119, 117, 55, 100, 73, 101, 66, 32, 59, 109, 110, 104, 58, 98, 65, 32, 115, 108, 117, 32, 116, 32, 87, 76, 32, 107, 101, 116, 65, 116, 108, 68, 110, 107, 59, 114, 78, 108, 85, 32, 73, 76, 87, 101, 87, 55, 32, 105, 105, 68, 119, 85, 108, 110, 114, 84, 101, 69, 105, 111, 108, 116, 116, 100, 89, 84, 32, 79, 116, 119, 69, 101, 68, 98, 85, 101, 32, 105, 32, 104, 32, 72, 78, 102, 114, 65, 102, 99, 32, 100, 86, 81, 100, 107, 32, 65, 114, 122, 84, 79, 76, 97, 114, 65, 32, 105, 102, 32, 107, 121, 101, 59, 32, 116, 78, 32, 115, 84, 76, 121, 115, 32, 111, 84, 100, 0};
const __constant__ U8 INITIAL_KEY[KEY_LEN] = { 24, 4, 25, 15, 25, 15, 25 };

__global__ void l2aKernel(U32 *outputBuffer) {
    int invocation_id = blockIdx.x * blockDim.x + threadIdx.x;
    U32 result = 0;

    short current_score;
    int data_index;
    int previous_printable_data_index;

    #pragma unroll
    for (int j = 0; j < ITERATIONS_PER_INVOCATION; j++) {
        /*
        Formatting checks:
        59 = ';'
        32 = ' '
        1) TODO:
        
        */
        current_score = 0;
        data_index = INITIAL_KEY[0];
        previous_printable_data_index = data_index;

        /*
        data_removed_count = [current iterations count]
        printable_data_index = data_index + data_removed_count
        key_index = [current iteration count] % [key length]
        */
        #pragma unroll
        for (int i = ACTUAL_DATA_SIZE - 1; i > ACTUAL_DATA_SIZE - L2A_SAFE_CHARS_COUNT; i--) {
            data_index = (data_index + INITIAL_KEY[(ACTUAL_DATA_SIZE - i) % KEY_LEN]) % i;
            current_score += (DATA4[previous_printable_data_index] > 96) & (DATA4[data_index + ACTUAL_DATA_SIZE - i] == 32);
            previous_printable_data_index = data_index + ACTUAL_DATA_SIZE - i;
        }
        result |= (current_score > PASSING_SCORE) * (1 << j);
    }

    outputBuffer[invocation_id] = result;
}


// Handles error in Debug mode, but ignores them in Release
inline hipError_t cudaUnwrap(hipError_t result) {
    #if defined(DEBUG) || defined(_DEBUG)
        if (result != cudaSuccess) {
            fprintf(stderr, "CUDA Runtime Error: %sn\n", cudaGetErrorString(result));
            exit(1);
        }
    #endif
    return result;
}

// Helper function for calling kernel
void l2aWithCuda(U32 *outputBuffer) {
    // Allocate initial key array
    // cudaUnwrap(cudaMalloc((void**)&initialKeyDevicePtr, KEY_LEN * sizeof(U8)));
    // cudaUnwrap(cudaMemcpy(initialKeyDevicePtr, initialKey, KEY_LEN * sizeof(U8), cudaMemcpyHostToDevice));
    // Launch kernel
    l2aKernel<<<BLOCK_COUNT, THREADS_COUNT>>>(outputBuffer);
    // Check for any errors after launch
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "l2aKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        // free(initialKeyDevicePtr);
        exit(1);
    }
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaUnwrap(hipDeviceSynchronize());
}


int main() {
    // Select GPU
    cudaUnwrap(hipSetDevice(0));

    const U8 INITIAL_KEY[KEY_LEN] = { 24, 4, 25, 15, 25, 15, 25 };
    // Allocating output buffer on unified memory
    U32* outputBufferHost;
    // TODO: flags, finish commit message
    cudaUnwrap(hipMallocManaged((void**)&outputBufferHost, OUTPUT_BUFFER_LEN * sizeof(U32)));

    // Messuaring execution time using cuda events
    // (See: https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/#using-cuda-gpu-timers)
    float exectuionTimeMs = 0.0;
    hipEvent_t executionStart, executionEnd;
    cudaUnwrap(hipEventCreate(&executionStart));
    cudaUnwrap(hipEventCreate(&executionEnd));

    cudaUnwrap(hipEventRecord(executionStart, 0));

    printf("Starting bruteforcer, pray to all gods that you know...\n");
    l2aWithCuda(outputBufferHost);

    cudaUnwrap(hipEventRecord(executionEnd, 0));
    // Execution time
    cudaUnwrap(hipEventSynchronize(executionEnd));
    cudaUnwrap(hipEventElapsedTime(&exectuionTimeMs, executionStart, executionEnd));
    printf("Elapsed: %f\n", exectuionTimeMs);

    cudaUnwrap(hipEventDestroy(executionStart));
    cudaUnwrap(hipEventDestroy(executionEnd));
    
    // LEVEL 4 OUTPUT: ifo llszu uyjx just kiddi
    for (int i = OUTPUT_BUFFER_LEN - 1; i < OUTPUT_BUFFER_LEN; i++) {
        printf("%u ", outputBufferHost[i]);
    }
    cudaUnwrap(hipFree(outputBufferHost));

    printf("\n");
    printf("\nDONE!\n");
    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaUnwrap(hipDeviceReset());
    return 0;
}
