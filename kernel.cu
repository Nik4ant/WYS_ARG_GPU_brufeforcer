﻿/*
Let's do some math:

THREADS_COUNT = 1024  // 1024 is limit on my GPU (can be higher?)
BLOCK_COUNT = 220  // amount of memory is the only limit here...

Each block has THREADS_COUNT threads.
One kernel (GPU device) invocation has BLOCK_COUNT * THREADS_COUNT parallel workers

NOTE: BLOCK_COUNT can be higher because right now my garbage code isn't super efficient, 
so I'm running out of memory. 


*/

#include "hip/hip_runtime.h"


#include <stdio.h>

// FUTURE TODO: Do all sorts of crazy optimazations (__restrict__, branchless and other crazy stuff)
// TODO: explain

typedef unsigned char U8;
typedef unsigned short U16;
typedef unsigned int U32;
typedef unsigned long long int U64;

const int THREADS_COUNT = 1024;  // 1024 <-- limit
const int BLOCK_COUNT = 220;  // 226 <-- UNSTABLE limit (not enough memory on the stack)

// TODO: replace some things with DEFINE?

// Bruteforcer params
constexpr auto MAX_SKIP = 26;
constexpr auto L2A_SAFE_CHARS_COUNT = 14;  // 14 for DATA5 using key of length 27 with maximum skips
constexpr auto ITERATIONS_PER_INVOCATION = 32;
constexpr auto KEY_LEN = 7;
constexpr auto PASSING_SCORE = 2;
// DATA params
constexpr auto ACTUAL_DATA_SIZE = 511;
constexpr auto ALIGNED_DATA_LEN = 512;
// TODO: test what happens without using aligned length...
const __constant__ U8 DATA4[ALIGNED_DATA_LEN] = { 73, 118, 84, 117, 105, 116, 110, 32, 116, 97, 116, 32, 32, 101, 72, 101, 110, 111, 69, 105, 32, 112, 103, 118, 105, 32, 73, 65, 84, 102, 32, 110, 65, 79, 105, 77, 32, 112, 76, 108, 32, 115, 115, 121, 71, 110, 101, 108, 110, 117, 32, 108, 40, 69, 101, 111, 32, 115, 101, 73, 116, 104, 72, 78, 99, 59, 89, 65, 82, 115, 76, 32, 111, 101, 101, 111, 69, 105, 108, 110, 66, 115, 116, 40, 32, 101, 108, 116, 104, 32, 100, 121, 108, 97, 114, 120, 116, 108, 83, 65, 80, 97, 32, 89, 98, 83, 112, 78, 82, 39, 85, 114, 115, 108, 67, 80, 116, 104, 59, 66, 32, 76, 105, 121, 82, 79, 115, 111, 77, 112, 115, 70, 32, 83, 114, 97, 115, 67, 73, 115, 115, 110, 32, 75, 78, 69, 65, 76, 104, 65, 84, 105, 104, 32, 111, 32, 32, 97, 116, 111, 119, 114, 115, 116, 122, 67, 110, 105, 115, 117, 104, 99, 65, 72, 75, 65, 109, 32, 32, 101, 99, 73, 82, 69, 102, 116, 78, 115, 85, 77, 105, 78, 110, 97, 110, 32, 72, 101, 111, 115, 105, 114, 104, 72, 83, 115, 59, 65, 100, 116, 117, 117, 78, 112, 111, 100, 112, 69, 111, 59, 112, 101, 121, 32, 114, 54, 84, 32, 41, 114, 115, 40, 105, 115, 103, 32, 116, 121, 65, 105, 59, 115, 115, 101, 59, 116, 70, 101, 67, 116, 117, 116, 69, 106, 32, 83, 111, 101, 119, 117, 111, 101, 32, 116, 77, 78, 82, 32, 66, 32, 83, 101, 100, 99, 108, 32, 67, 32, 75, 120, 101, 101, 67, 79, 120, 97, 105, 59, 32, 116, 110, 105, 116, 76, 108, 76, 121, 32, 116, 101, 116, 110, 99, 85, 32, 79, 79, 32, 105, 106, 119, 69, 105, 67, 58, 98, 55, 79, 59, 101, 85, 105, 116, 104, 120, 32, 86, 79, 111, 100, 65, 114, 32, 41, 84, 117, 32, 120, 98, 115, 99, 58, 111, 110, 68, 32, 78, 110, 32, 116, 112, 115, 111, 116, 32, 75, 83, 101, 111, 59, 97, 101, 41, 68, 77, 69, 119, 50, 122, 32, 116, 102, 32, 69, 121, 102, 111, 116, 100, 97, 110, 114, 65, 101, 101, 116, 119, 117, 55, 100, 73, 101, 66, 32, 59, 109, 110, 104, 58, 98, 65, 32, 115, 108, 117, 32, 116, 32, 87, 76, 32, 107, 101, 116, 65, 116, 108, 68, 110, 107, 59, 114, 78, 108, 85, 32, 73, 76, 87, 101, 87, 55, 32, 105, 105, 68, 119, 85, 108, 110, 114, 84, 101, 69, 105, 111, 108, 116, 116, 100, 89, 84, 32, 79, 116, 119, 69, 101, 68, 98, 85, 101, 32, 105, 32, 104, 32, 72, 78, 102, 114, 65, 102, 99, 32, 100, 86, 81, 100, 107, 32, 65, 114, 122, 84, 79, 76, 97, 114, 65, 32, 105, 102, 32, 107, 121, 101, 59, 32, 116, 78, 32, 115, 84, 76, 121, 115, 32, 111, 84, 100, 0};

__global__ void l2aKernel(U32* output_buffer, const U8* __restrict__ initial_key)
{
    auto invocation_id = blockIdx.x * blockDim.x + threadIdx.x;
    U32 result = 0;

    short current_score;
    int data_index;
    int previous_printable_data_index;

    // Works when iterations count is compile known value
    /*
    data_removed_count = [current iterations count]
    printable_data_index = data_index + data_removed_count
    key_index = [current iteration count] % [key length]
    */
    
    // WARNING: NEED TO IMPLEMENT PROPER KEY ITERATIONS AND SAFE l2a bounds
    // TODO: test manual unrolling to see if it does what intended
    #pragma unroll
    for (int j = 0; j < ITERATIONS_PER_INVOCATION; j++) {
        /*
        Formatting checks:
        59 = ';'
        32 = ' '
        1) TODO:
        
        */
        current_score = 0;
        // 1 step of the decryption
        data_index = initial_key[0];
        previous_printable_data_index = data_index;
        current_score -= (DATA4[data_index] == 32 | DATA4[data_index] == 59);
        // 2..Nth steps
        #pragma unroll
        for (int i = ACTUAL_DATA_SIZE - 1; i > ACTUAL_DATA_SIZE - L2A_SAFE_CHARS_COUNT; i--) {
            // debug only
            // output_buffer[ACTUAL_DATA_SIZE - i] = DATA4[data_index + ACTUAL_DATA_SIZE - i];

            data_index = (data_index + initial_key[(ACTUAL_DATA_SIZE - i) % KEY_LEN]) % i;
            // Characters before 97 aren't lowercase english laters
            current_score += ((DATA4[previous_printable_data_index] > 96) & (DATA4[data_index + ACTUAL_DATA_SIZE - i] == 32 | DATA4[data_index + ACTUAL_DATA_SIZE - i] == 59));
            previous_printable_data_index = data_index + ACTUAL_DATA_SIZE - i;
        }
        result |= (current_score >= PASSING_SCORE) * (1 << j);
    }

    output_buffer[invocation_id] = result;
}


hipError_t l2aWithCuda(U32* output_buffer, const U8* __restrict__ initial_key, const size_t output_buffer_size);


int main() {
    const unsigned int OUTPUT_BUFFER_LEN = THREADS_COUNT * BLOCK_COUNT;
    const U8 INITIAL_KEY[KEY_LEN] = { 24, 4, 25, 15, 25, 15, 25 };

    U32 output_buffer[OUTPUT_BUFFER_LEN] = { 0 };
    
    // Messuaring execution time using cuda events
    // (See: https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/#using-cuda-gpu-timers)
    float exectuionTimeMs = 0.0;
    hipEvent_t executionStart, executionEnd;
    hipEventCreate(&executionStart);
    hipEventCreate(&executionEnd);

    // Running CUDA
    hipEventRecord(executionStart, 0);
    hipError_t cudaStatus = l2aWithCuda(output_buffer, INITIAL_KEY, OUTPUT_BUFFER_LEN * sizeof(U32));
    hipEventRecord(executionEnd, 0);
    // Execution time
    hipEventSynchronize(executionEnd);
    hipEventElapsedTime(&exectuionTimeMs, executionStart, executionEnd);
    printf("Elapsed: %f ms\n", exectuionTimeMs);
    hipEventDestroy(executionStart);
    hipEventDestroy(executionEnd);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "l2aWithCuda failed!\n");
        return 1;
    }
    
    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    
    // LEVEL 4 OUTPUT: ifo llszu uyjx just kiddi
    for (int i = OUTPUT_BUFFER_LEN - 1; i < OUTPUT_BUFFER_LEN; i++) {
        printf("%u ", output_buffer[i]);
    }
    printf("\n");

    printf("\nDONE!\n");
    // printf("Press ENTER key to exit\n");
    // getchar();
    return 0;
}


// Helper function for using CUDA
hipError_t l2aWithCuda(U32* output_buffer, const U8* __restrict__ initial_key, const size_t output_buffer_size)
{
    U8* device_ptr_key = 0;
    U32* device_ptr_output_buffer = 0;
    hipError_t cudaStatus;

    // Select GPU
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocating GPU buffers.
    // Output buffer
    cudaStatus = hipMalloc((void**)&device_ptr_output_buffer, output_buffer_size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! (output buffer)");
        goto Error;
    }
    // Key buffer
    cudaStatus = hipMalloc((void**)&device_ptr_key, KEY_LEN * sizeof(U8));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! (key data)");
        goto Error;
    }

    // Coping buffers from host (CPU) to GPU.
    // Key data
    cudaStatus = hipMemcpy(device_ptr_key, initial_key, KEY_LEN * sizeof(U8), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! (key data)");
        goto Error;
    }

    // Launching kernel
    l2aKernel<<<BLOCK_COUNT, THREADS_COUNT>>>(device_ptr_output_buffer, device_ptr_key);

    // Check for any errors after launch
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output from GPU to host (CPU)
    cudaStatus = hipMemcpy(output_buffer, device_ptr_output_buffer, output_buffer_size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(device_ptr_key);
    hipFree(device_ptr_output_buffer);

    return cudaStatus;
}

