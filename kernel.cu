﻿#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <cmath>

typedef unsigned char U8;
typedef unsigned short U16;
typedef unsigned int U32;
typedef unsigned long long int U64;

// TODO: lower threads count could perform better overall? Test later
const int DEFAULT_THREADS_COUNT = 256;  // 1024 <-- limit (can be changed, but that's probably a bad idea)
const int DEFAULT_BLOCKS_COUNT = 1024;  // 4096 * 256 <-- stable. AMOUNT OF MEMORY IS THE ONLY LIMIT

// Bruteforcer params
constexpr auto MAX_SKIP = 26;
constexpr auto L2A_SAFE_CHARS_COUNT = 14;  // 14 for DATA5 using key of length 27 with maximum skips
constexpr auto ITERATIONS_PER_INVOCATION = 32;
constexpr auto KEY_LEN = 7;
constexpr auto PASSING_SCORE = 1;  // used with >
// DATA params
constexpr auto ACTUAL_DATA_SIZE = 511;
constexpr auto ALIGNED_DATA_LEN = 512;

const __constant__ U8 DATA4[ALIGNED_DATA_LEN] = { 73, 118, 84, 117, 105, 116, 110, 32, 116, 97, 116, 32, 32, 101, 72, 101, 110, 111, 69, 105, 32, 112, 103, 118, 105, 32, 73, 65, 84, 102, 32, 110, 65, 79, 105, 77, 32, 112, 76, 108, 32, 115, 115, 121, 71, 110, 101, 108, 110, 117, 32, 108, 40, 69, 101, 111, 32, 115, 101, 73, 116, 104, 72, 78, 99, 59, 89, 65, 82, 115, 76, 32, 111, 101, 101, 111, 69, 105, 108, 110, 66, 115, 116, 40, 32, 101, 108, 116, 104, 32, 100, 121, 108, 97, 114, 120, 116, 108, 83, 65, 80, 97, 32, 89, 98, 83, 112, 78, 82, 39, 85, 114, 115, 108, 67, 80, 116, 104, 59, 66, 32, 76, 105, 121, 82, 79, 115, 111, 77, 112, 115, 70, 32, 83, 114, 97, 115, 67, 73, 115, 115, 110, 32, 75, 78, 69, 65, 76, 104, 65, 84, 105, 104, 32, 111, 32, 32, 97, 116, 111, 119, 114, 115, 116, 122, 67, 110, 105, 115, 117, 104, 99, 65, 72, 75, 65, 109, 32, 32, 101, 99, 73, 82, 69, 102, 116, 78, 115, 85, 77, 105, 78, 110, 97, 110, 32, 72, 101, 111, 115, 105, 114, 104, 72, 83, 115, 59, 65, 100, 116, 117, 117, 78, 112, 111, 100, 112, 69, 111, 59, 112, 101, 121, 32, 114, 54, 84, 32, 41, 114, 115, 40, 105, 115, 103, 32, 116, 121, 65, 105, 59, 115, 115, 101, 59, 116, 70, 101, 67, 116, 117, 116, 69, 106, 32, 83, 111, 101, 119, 117, 111, 101, 32, 116, 77, 78, 82, 32, 66, 32, 83, 101, 100, 99, 108, 32, 67, 32, 75, 120, 101, 101, 67, 79, 120, 97, 105, 59, 32, 116, 110, 105, 116, 76, 108, 76, 121, 32, 116, 101, 116, 110, 99, 85, 32, 79, 79, 32, 105, 106, 119, 69, 105, 67, 58, 98, 55, 79, 59, 101, 85, 105, 116, 104, 120, 32, 86, 79, 111, 100, 65, 114, 32, 41, 84, 117, 32, 120, 98, 115, 99, 58, 111, 110, 68, 32, 78, 110, 32, 116, 112, 115, 111, 116, 32, 75, 83, 101, 111, 59, 97, 101, 41, 68, 77, 69, 119, 50, 122, 32, 116, 102, 32, 69, 121, 102, 111, 116, 100, 97, 110, 114, 65, 101, 101, 116, 119, 117, 55, 100, 73, 101, 66, 32, 59, 109, 110, 104, 58, 98, 65, 32, 115, 108, 117, 32, 116, 32, 87, 76, 32, 107, 101, 116, 65, 116, 108, 68, 110, 107, 59, 114, 78, 108, 85, 32, 73, 76, 87, 101, 87, 55, 32, 105, 105, 68, 119, 85, 108, 110, 114, 84, 101, 69, 105, 111, 108, 116, 116, 100, 89, 84, 32, 79, 116, 119, 69, 101, 68, 98, 85, 101, 32, 105, 32, 104, 32, 72, 78, 102, 114, 65, 102, 99, 32, 100, 86, 81, 100, 107, 32, 65, 114, 122, 84, 79, 76, 97, 114, 65, 32, 105, 102, 32, 107, 121, 101, 59, 32, 116, 78, 32, 115, 84, 76, 121, 115, 32, 111, 84, 100, 0};
const __constant__ U8 INITIAL_KEY[KEY_LEN] = { 24, 4, 25, 15, 25, 15, 25 };
// TODO: generate later at compile time by using KEY_LEN param
const __constant__ U8 KEY_MODULO_LOOKUP_TABLE[L2A_SAFE_CHARS_COUNT] = {
    0, 1, 2, 3, 4, 5, 6, 0, 1, 2, 3, 4, 5, 6
};
// TODO: generate later at compile time by using ITERATIONS_PER_INVOCATION param
const __constant__ U32 BITS_SHIFTING_LOOKUP_TABLE[ITERATIONS_PER_INVOCATION] = {
    1 << 0, 1 << 1, 1 << 2, 1 << 3, 1 << 4, 1 << 5, 1 << 6, 1 << 7, 1 << 8, 1 << 9, 1 << 10, 1 << 11, 
    1 << 12, 1 << 13, 1 << 14, 1 << 15, 1 << 16, 1 << 17, 1 << 18, 1 << 19, 1 << 20, 1 << 21, 1 << 22, 
    1 << 23, 1 << 24, 1 << 25, 1 << 26, 1 << 27, 1 << 28, 1 << 29, 1 << 30, 1 << 31
};

__global__ void l2aKernel(U32 *outputBuffer) {
    int invocation_id = blockIdx.x * blockDim.x + threadIdx.x;
    U32 result = 0;

    U8 current_score;
    int data_index;
    int previous_printable_data_index;

    #pragma unroll (ITERATIONS_PER_INVOCATION)
    for (int j = 0; j < ITERATIONS_PER_INVOCATION; j++) {
        current_score = 0;
        data_index = INITIAL_KEY[0];
        previous_printable_data_index = data_index;

        /*
        data_removed_count = [current iterations count]
        printable_data_index = data_index + data_removed_count
        key_index = [current iteration count] % [key length]
        */
        #pragma unroll
        for (int i = ACTUAL_DATA_SIZE - 1; i > ACTUAL_DATA_SIZE - L2A_SAFE_CHARS_COUNT; i--) {
            data_index = (data_index + INITIAL_KEY[KEY_MODULO_LOOKUP_TABLE[(ACTUAL_DATA_SIZE - i)]]) % i;
            current_score += (DATA4[previous_printable_data_index] > 96) & (DATA4[data_index + ACTUAL_DATA_SIZE - i] == 32);
            previous_printable_data_index = data_index + ACTUAL_DATA_SIZE - i;
        }
        result |= (current_score > PASSING_SCORE) * BITS_SHIFTING_LOOKUP_TABLE[j];
    }

    outputBuffer[invocation_id] = result;
}


// Handles error in Debug mode, but ignores them in Release
inline hipError_t cudaUnwrap(hipError_t result) {
    #if defined(DEBUG) || defined(_DEBUG)
        if (result != cudaSuccess) {
            fprintf(stderr, "CUDA Runtime Error: %sn\n", cudaGetErrorString(result));
            exit(1);
        }
    #endif
    return result;
}

// Helper function for calling kernel
void l2aWithCuda(U32 *outputBuffer, unsigned long threads, unsigned long blocks) {
    // Allocate initial key array
    // cudaUnwrap(cudaMalloc((void**)&initialKeyDevicePtr, KEY_LEN * sizeof(U8)));
    // cudaUnwrap(cudaMemcpy(initialKeyDevicePtr, initialKey, KEY_LEN * sizeof(U8), cudaMemcpyHostToDevice));
    // Launch kernel
    l2aKernel<<<blocks, threads>>>(outputBuffer);
    // Check for any errors after launch
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "l2aKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        // free(initialKeyDevicePtr);
        exit(1);
    }
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaUnwrap(hipDeviceSynchronize());
}

//
//
//
//
// NOTE: THIS GUY IS GENIUS; USE HIS TIPS: http://impact.crhc.illinois.edu/shared/report/phd-thesis-shane-ryoo.pdf
// TODO: open intal-D music in the backround at the beggining xD
//
//
//
int main(int argc, char **argv) {
    unsigned long threads_count = DEFAULT_THREADS_COUNT;
    unsigned long blocks_count = DEFAULT_BLOCKS_COUNT;
    if (argc == 3) {
        // argv[0] is executable path (not sure if it's always the case)
        printf("Detected 2 args; `%s` threads and `%s` blocks\n", argv[1], argv[2]);
        // There is no such thing as safety, only hope...
        threads_count = strtoul(argv[1], &argv[1], 10);
        blocks_count = strtoul(argv[2], &argv[2], 10);
    } else {
        printf("No args for threads count and blocks count were detected; Using default values...\n");
    }

    U32 output_buffer_len = threads_count * blocks_count;

    // Select GPU
    cudaUnwrap(hipSetDevice(0));

    // Allocating output buffer
    U32* outputBufferHost;
    // TODO: benchmark different variants
    cudaUnwrap(hipMallocManaged((void**)&outputBufferHost, threads_count * blocks_count * sizeof(U32)));
    // TODO: async prefetching + double buffering
    #pragma unroll
    for (U32 i = 0; i < output_buffer_len; i++) {
        outputBufferHost[i] = i;
    }
    
    // FIXME: THIS NUMBER IS WRONG; smth wrong with the types
    long double cudaCallsCount = ceil(powl(MAX_SKIP, KEY_LEN) / (output_buffer_len * ITERATIONS_PER_INVOCATION));
    printf("Starting bruteforcer, pray to all gods that you know...\n");
    // printf("(NOT WORKING )Expected amount of calls to cuda kernel = %Lf\n", cudaCallsCount);

    // Messuaring execution time using cuda events
    // (See: https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/#using-cuda-gpu-timers)
    float exectuionTimeMs = 0.0;
    hipEvent_t executionStart, executionEnd;
    cudaUnwrap(hipEventCreate(&executionStart));
    cudaUnwrap(hipEventCreate(&executionEnd));
    cudaUnwrap(hipEventRecord(executionStart, 0));

    // TODO: manual unrolling? (this tag doesn't seem to work here)
    #pragma unroll
    for (int cudaCallIndex = 0; cudaCallIndex < 1; cudaCallIndex++) {
        l2aWithCuda(outputBufferHost, threads_count, blocks_count);
        // Step 2. Check keys that produce something that doesn't classified as giberish
        #pragma unroll
        for (U32 i = 0; i < output_buffer_len; i++) {
            if (outputBufferHost[i] != 0) {
                // TODO: find a way to turn that into a switch statement (since there are only 32 bits)
                for (int bitPosition = 0; bitPosition < 32; bitPosition++) {
                    if ((outputBufferHost[i] >> bitPosition) & 1) {
                        // printf("Found a key: TODO:");
                    }
                }
                // printf("\n");
                break;
            }
        }
    }
    cudaUnwrap(hipEventRecord(executionEnd, 0));
    // Execution time
    cudaUnwrap(hipEventSynchronize(executionEnd));
    cudaUnwrap(hipEventElapsedTime(&exectuionTimeMs, executionStart, executionEnd));
    printf("Elapsed: %fms\n", exectuionTimeMs);

    cudaUnwrap(hipEventDestroy(executionStart));
    cudaUnwrap(hipEventDestroy(executionEnd));
    
    // LEVEL 4 OUTPUT: ifo llszu uyjx just kiddi
    for (U32 i = output_buffer_len - 1; i < output_buffer_len; i++) {
        printf("%u ", outputBufferHost[i]);
    }
    cudaUnwrap(hipFree(outputBufferHost));

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaUnwrap(hipDeviceReset());
    return 0;
}
